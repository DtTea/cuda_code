#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <sys/time.h>


#include < time.h >
#include <windows.h> //I've ommited this line.
#if defined(_MSC_VER) || defined(_MSC_EXTENSIONS)
  #define DELTA_EPOCH_IN_MICROSECS  11644473600000000Ui64
#else
  #define DELTA_EPOCH_IN_MICROSECS  11644473600000000ULL
#endif
 
struct timezone 
{
  int  tz_minuteswest; /* minutes W of Greenwich */
  int  tz_dsttime;     /* type of dst correction */
};
 
int gettimeofday(struct timeval *tv, struct timezone *tz)
{
  FILETIME ft;
  unsigned __int64 tmpres = 0;
  static int tzflag;
 
  if (NULL != tv)
  {
    GetSystemTimeAsFileTime(&ft);
 
    tmpres |= ft.dwHighDateTime;
    tmpres <<= 32;
    tmpres |= ft.dwLowDateTime;
 
    /*converting file time to unix epoch*/
    tmpres -= DELTA_EPOCH_IN_MICROSECS; 
    tmpres /= 10;  /*convert into microseconds*/
    tv->tv_sec = (long)(tmpres / 1000000UL);
    tv->tv_usec = (long)(tmpres % 1000000UL);
  }
 
  if (NULL != tz)
  {
    if (!tzflag)
    {
      _tzset();
      tzflag++;
    }
    tz->tz_minuteswest = _timezone / 60;
    tz->tz_dsttime = _daylight;
  }
 
  return 0;
}



double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}
void initCpu(float *hostA, float *hostB, int n)
{
    for (int i = 0; i < n; i++)
    {
        hostA[i] = 1;
        hostB[i] = 1;
    }
}
void addCpu(float *hostA, float *hostB, float *hostC, int n)
{
    for (int i = 0; i < n; i++)
    {
        hostC[i] = hostA[i] + hostB[i];
    }
}
__global__ void addKernel(float *deviceA, float *deviceB, float *deviceC, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x; // 计算全局索引
    if (index < n)
    {
        deviceC[index] = deviceA[index] + deviceB[index];
    }
}
int main()
{
    float *hostA, *hostB, *hostC, *serialC;
    int n = 102400;

    hostA = (float *)malloc(n * sizeof(float));
    hostB = (float *)malloc(n * sizeof(float));
    hostC = (float *)malloc(n * sizeof(float));
    serialC = (float *)malloc(n * sizeof(float));
    initCpu(hostA, hostB, n);
    double stC, elaC;
    stC = get_walltime();
    addCpu(hostA, hostB, serialC, n);
    elaC = get_walltime() - stC;
    double st, ela;
    st = get_walltime();

    float *dA, *dB, *dC;
    hipMalloc((void **)&dA, n * sizeof(float));
    hipMalloc((void **)&dB, n * sizeof(float));
    hipMalloc((void **)&dC, n * sizeof(float));

    hipMemcpy(dA, hostA, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hostB, n * sizeof(float), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    int BLOCK_DIM = 1024;
    int num_block_x = n / BLOCK_DIM;
    int num_block_y = 1;
    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(BLOCK_DIM, 1, 1);
    addKernel<<<grid_dim, block_dim>>>(dA, dB, dC, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time
    hipMemcpy(hostC, dC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    ela = get_walltime() - st;
    printf("n = %d: \n CPU use time:%.4f\n GPU use time:%.4f\n kernel time:%.4f\n", n, elaC, ela, ker_time / 1000.0);
    free(hostA);
    free(hostB);
    free(hostC);
    free(serialC);
    return 0;
}